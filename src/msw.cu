#include "hip/hip_runtime.h"
#include <bit> // bit_width
#include <cstddef> // size_t
#include <cstdint> // uintmax_t
#include <iostream>

#include <stdio.h>

#include "cuda_utils.cuh"

/**
 * @brief CUDA kernel that implements the first phase of the square root decomposition.
 * @details This kernel performs two critical operations:
 *   1. Computes the XOR (as inequality check) between corresponding elements of two arrays
 *   2. Maps each mismatch to its respective sqrt(n)-sized chunk in the signature array
 *
 * Each thread processes multiple elements based on its thread ID. When a mismatch
 * is found, the corresponding chunk in sig_out is marked with a 1.
 */
__global__ void par_sig_sqrt(const uintmax_t * const p1, uintmax_t *p2, uintmax_t *sig_out, size_t n) {
	// Calculate number of chunks, each of size sqrt(n)
	size_t num_sqrt = std::sqrt(n - 1) + 1;

	// Each thread processes elements at positions (threadIdx + k*blockDim*gridDim)
	for (auto i = get_tid(); i < n; i += get_num_threads()) {
		// Store XOR result in p2 - non-zero values indicate mismatches
		p2[i] = p1[i] != p2[i];

		// If there's a mismatch, mark its chunk in the signature array
		if (p2[i] != 0)
			sig_out[i / num_sqrt] = 1;
	}
}

/**
 * @brief Parallel kernel to find the leftmost non-zero element in an array.
 * @details Uses a tournament-style approach where each pair of elements is compared,
 * and if a left element (lower index) is non-zero, it "eliminates" the right element
 * by setting it to zero. After all comparisons, only the leftmost non-zero element remains.
 * This kernel achieves O(n²) work in O(1) span when sufficient threads are available.
 *
 * The algorithm distributes all n-choose-2 pairwise comparisons across available threads.
 */
__global__ void naive_leftmost_prisoner(uintmax_t *p, size_t n)
{
	// Calculate total number of pairwise comparisons needed
	size_t n_choose_2 = n * (n - 1) / 2;
	
	// Distribute comparisons evenly across threads
	size_t num_computations = (n_choose_2 - 1) / get_num_threads() + 1;

	// Calculate starting comparison index for this thread
	auto i = get_tid() * num_computations;
	
	// Convert linear index i to pair (a,b) using inverse of triangular number formula
	// This maps from a 1D index to a 2D comparison between elements at positions a and b
	size_t b = (1 + std::sqrt(1 + 8 * i)) / 2;  // Row index calculation
	size_t a = i - b * (b - 1) / 2;             // Column index calculation
	
	// Process this thread's allocated comparisons
	while (num_computations-- && b < n)
	{
		// If the left element is non-zero, eliminate the right element
		if (p[a]) p[b] = 0;

		// Move to next comparison pair
		++a;

		// If we reach the end of a row, move to the next row
		if (a == b)
		{
			a = 0;  // Start at the leftmost element
			++b;    // Move to the next row
		}
	}
}

/**
 * @brief Parallel kernel to identify and record the index of the first non-zero element.
 * @details After the naive_leftmost_prisoner kernel has eliminated all non-leftmost elements,
 * this kernel scans the array to find the remaining non-zero element and records its index.
 * Multiple threads may write to the same location, but they'll all write the same value
 * since only one element should remain non-zero after the elimination process.
 * 
 * This kernel is used in both phases of the square root algorithm:
 * 1. To find which sqrt(n)-sized chunk contains the first mismatch
 * 2. To find the exact position within that chunk
 */
__global__ void par_get_section_diff(uintmax_t *tree, uintmax_t *section, size_t n) 
{
	// Each thread checks a subset of elements
	for (auto i = get_tid(); i < n; i += get_num_threads())
		// If we find a non-zero element, it must be the leftmost one after elimination
		if (tree[i] != 0)
			*section = i;  // Record its index as the result
}

size_t seq_find_mismatch(const uintmax_t * const arr1, const uintmax_t * const arr2, size_t size) 
{
	for (size_t i = 0; i < size; ++i)
		if (arr1[i] != arr2[i])
			return i;

	return size;
}

// assume d_a is already populated on the device
// assume large block has size big enough to hold b and signature array
// that size should be ~n + sqrt(n) + 2
size_t par_find_mismatch(const uintmax_t * const d_a, const uintmax_t * const b, uintmax_t *d_large_block, size_t n)
{
	// Copy the second array to device memory using pre-allocated buffer
	uintmax_t *d_b = copy_to_device(d_large_block, b, n);

	// Set up memory pointers within the large block:
	// - d_section: single value to store result
	// - d_sqrt: array of size sqrt(n) to track which chunks have mismatches
	uintmax_t *d_section = d_b + n;
	uintmax_t *d_sqrt = d_section + 1;
	size_t num_sqrt = std::sqrt(n - 1) + 1;  // Calculate number of sqrt-sized chunks

	// Initialize the section result to max value (no match found yet)
	memset_within_device(d_section, 1, 0xFF);

	// Initialize the sqrt array to zeros (no mismatches found yet)
	memset_within_device(d_sqrt, num_sqrt, 0);

	// PHASE 1: Find which sqrt(n)-sized chunk has the first mismatch
	// For each element, compute XOR of the two arrays and mark its chunk in d_sqrt if mismatch
	par_sig_sqrt<<<BLOCKS, THREADS>>>(d_a, d_b, d_sqrt, n);

	// Find the first non-zero chunk (chunk with a mismatch) in parallel
	naive_leftmost_prisoner<<<BLOCKS, THREADS>>>(d_sqrt, num_sqrt);

	// Extract the index of the first chunk with a mismatch
	par_get_section_diff<<<BLOCKS, THREADS>>>(d_sqrt, d_section, num_sqrt);

	// Copy the result back to host memory
	uintmax_t *h_section = copy_from_device(d_section, 1);
	size_t section = *h_section;
	
	// If no mismatch found, return n (indicating arrays are identical)
	if (section == std::numeric_limits<uintmax_t>::max()) {
		delete[] h_section;
		return n;
	}

	// Reset the section result for the second phase
	memset_within_device(d_section, 1, 0xFF);

	// PHASE 2: Find the exact mismatch position within the identified chunk
	// Calculate the offset and size of the identified chunk
	size_t section_offset = section * num_sqrt;
	size_t section_end = std::min(n, section_offset + num_sqrt);
	uintmax_t *d_sig_section = d_b + section_offset;
	size_t num_words = section_end - section_offset;

	// Find the first non-zero word within the chunk (containing XOR results)
	naive_leftmost_prisoner<<<BLOCKS, THREADS>>>(d_sig_section, num_words);

	// Extract the index of the first mismatching word within the chunk
	par_get_section_diff<<<BLOCKS, THREADS>>>(d_sig_section, d_section, num_words);

	// Copy the result back to host memory
	h_section = copy_from_device(d_section, 1);
	size_t result = *h_section;

	delete[] h_section;

	// Return the global index of the first mismatch (chunk offset + position within chunk)
	return section_offset + result;
}

uintmax_t* alloc_large_block_to_device(size_t n)
{
	size_t large_block_size = n + std::sqrt(n - 1) + 2;

	return alloc_to_device<uintmax_t>(large_block_size);
}

size_t par_find_mismatch(const uintmax_t * const a, const uintmax_t * const b, size_t n) 
{
	uintmax_t *d_a = copy_to_device(a, n);
	uintmax_t *d_large_block = alloc_large_block_to_device(n);

	size_t result = par_find_mismatch(d_a, b, d_large_block, n);

	device_free(d_a);
	device_free(d_large_block);

	return result;
}
