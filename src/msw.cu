#include "hip/hip_runtime.h"
#include <bit> // bit_width
#include <cstddef> // size_t
#include <cstdint> // uintmax_t
#include <iostream>

#include <stdio.h>

#include "utility.cuh"

// TODO: Naming conventions in general suck, should fix

/*
__host__ inline size_t host_fast_log2(size_t n) 
{
	return std::bit_width(n) - 1;
}

__host__ inline size_t host_fast_log2_ceil(size_t n) 
{
	return std::bit_width(n - 1);
}

__device__ __forceinline__ size_t device_fast_log2(size_t n) 
{
	return std::numeric_limits<size_t>::digits - 1 - __clzll(n);
}

__device__ __forceinline__ size_t device_fast_log2_ceil(size_t n) 
{
	return std::numeric_limits<size_t>::digits - __clzll(n - 1);
}
*/

__device__ __forceinline__ size_t device_get_ancestor(size_t i, size_t height) 
{
	return ((i + 1) >> height) - 1;
}

__global__ void par_xor_sig(const uintmax_t * const p1, uintmax_t *p2, uintmax_t *sig_out, size_t n) 
{
	for (auto i = get_tid(); i < n; i += get_num_threads()) 
	{
		p2[i] = p1[i] ^ p2[i];

		if (p2[i] != 0)
			sig_out[i / device_fast_log2(n)] = 1;
	}
}

__global__ void par_poptree(uintmax_t *tree, size_t n) 
{
	auto num_ancestors = device_fast_log2_ceil(n);
	auto leaf_start = (1uLL << num_ancestors) - 1; // there are 2^num_ancestors - 1 internal nodes

	for (auto i = get_tid(); i < n * num_ancestors; i += get_num_threads()) 
	{
		auto leaf_index = i / num_ancestors;
		auto leaf_node = leaf_start + leaf_index;

		if (tree[leaf_node] == 0)
			continue;

		auto height_from_bottom = (i % num_ancestors) + 1;

		tree[device_get_ancestor(leaf_node, height_from_bottom)] = 1;
	}
}

__global__ void par_mark_msw(uintmax_t *tree, uintmax_t n) 
{
	auto num_ancestors = device_fast_log2_ceil(n);
	auto leaf_start = (1uLL << num_ancestors) - 1; // there are 2^num_ancestors - 1 internal nodes

	for (auto i = get_tid(); i < n * num_ancestors; i += get_num_threads()) 
	{
		auto leaf_index = i / num_ancestors;
		auto leaf_node = leaf_start + leaf_index;

		if (tree[leaf_node] == 0)
			continue;

		auto height_from_bottom = (i % num_ancestors) + 1;
		auto ancestor = device_get_ancestor(leaf_node, height_from_bottom);
		auto ancestor_left_child = (ancestor << 1) + 1;

		if (tree[ancestor_left_child] == 0)
			continue;

		auto one_lower_ancestor = device_get_ancestor(leaf_node, height_from_bottom - 1);

		if (one_lower_ancestor == ancestor_left_child)
			continue;

		tree[leaf_node] = 0;
	}
}

__global__ void par_get_section_diff(uintmax_t *tree, uintmax_t *section, size_t n) 
{
	for (auto i = get_tid(); i < n; i += get_num_threads())
		if (tree[i] != 0)
			*section = i;
}

size_t seq_find_mismatch(const uintmax_t * const arr1, const uintmax_t * const arr2, size_t size) 
{
	for (size_t i = 0; i < size; ++i)
		if (arr1[i] != arr2[i])
			return i;

	return size;
}

size_t seq_find_msw(const uintmax_t * const words, size_t n)
{
	for (size_t i = 0; i < n; ++i)
		if (words[i] != 0)
			return i;

	return n;
}

size_t par_find_msw(uintmax_t *d_tree, uintmax_t *d_section, size_t num_leaves)
{
	size_t padded_num_leaves = 1 << host_fast_log2_ceil(num_leaves);
	uintmax_t *leaves = d_tree + padded_num_leaves - 1;

	memset_within_device(d_section, 1, 0xFF);

	par_poptree<<<BLOCKS, THREADS>>>(d_tree, num_leaves);

	par_mark_msw<<<BLOCKS, THREADS>>>(d_tree, num_leaves);

	par_get_section_diff<<<BLOCKS, THREADS>>>(leaves, d_section, num_leaves);

	uintmax_t *h_section = copy_from_device(d_section, 1);
	size_t section = *h_section;

	delete[] h_section;

	if (section == std::numeric_limits<uintmax_t>::max())
		return num_leaves;

	return section;
}

// assume d_a and d_b are already populated
// assume d_tree and d_section are already allocated
size_t par_find_mismatch(const uintmax_t * const d_a, uintmax_t *d_b, uintmax_t *d_tree, uintmax_t *d_section, size_t n)
{
	std::cout << "par_find_mismatch" << std::endl;
	std::cout << n << std::endl;
	size_t num_leaves_per_section = host_fast_log2(n);
	size_t num_leaves = (n + num_leaves_per_section - 1) / num_leaves_per_section;
	size_t padded_num_leaves = 1 << host_fast_log2_ceil(num_leaves);
	uintmax_t *leaves = d_tree + padded_num_leaves - 1;

	memset_within_device(d_tree, num_leaves + padded_num_leaves - 1, 0);

	std::cout << "par_find_mismatch 1" << std::endl;

	// stores XOR in d_b
	par_xor_sig<<<BLOCKS, THREADS>>>(d_a, d_b, leaves, n);

	uintmax_t *h_leaves = copy_from_device(leaves, padded_num_leaves);
	std::cout << "leaves: ";
	for (size_t i = 0; i < padded_num_leaves; ++i)
		std::cout << h_leaves[i] << " ";

	std::cout << std::endl;

	std::cout << "par_find_mismatch 2" << std::endl;

	size_t h_section = par_find_msw(d_tree, d_section, num_leaves);
	if (h_section == num_leaves)
		return n;

	std::cout << "par_find_mismatch 3" << std::endl;

	size_t xor_offset = h_section * num_leaves_per_section;
	size_t xor_end = std::min(n, xor_offset + num_leaves_per_section);
	uintmax_t *xor_section = d_b + xor_offset;
	size_t num_words = xor_end - xor_offset;

	// print all the above variables, very pretty, fancy message

	std::cout << "h_section: " << h_section << std::endl;
	std::cout << "num_leaves_per_section: " << num_leaves_per_section << std::endl;
	std::cout << "xor_offset: " << xor_offset << std::endl;
	std::cout << "xor_end: " << xor_end << std::endl;
	std::cout << "num_words: " << num_words << std::endl;


	uintmax_t *h_xor_section = copy_from_device(xor_section, num_words);

	std::cout << "par_find_mismatch 4" << std::endl;

	size_t result = seq_find_msw(h_xor_section, num_words);
	delete[] h_xor_section;

	std::cout << "par_find_mismatch end" << std::endl;

	return xor_offset + result;
}

// assume d_a is already populated
// assume large block has size big enough to hold b, tree, and section (1)
// that size should be ~n + 2 * (n / log2(n)) + 1
size_t par_find_mismatch(const uintmax_t * const d_a, const uintmax_t * const b, uintmax_t *d_large_block, size_t n)
{
	uintmax_t *d_b = copy_to_device(d_large_block, b, n);
	uintmax_t *d_section = d_b + n;
	uintmax_t *d_tree = d_section + 1;

	return par_find_mismatch(d_a, d_b, d_tree, d_section, n);
}

uintmax_t* alloc_large_block_to_device(size_t n)
{
	size_t num_leaves_per_section = host_fast_log2(n);
	size_t num_leaves = (n + num_leaves_per_section - 1) / num_leaves_per_section;
	size_t padded_num_leaves = 1 << host_fast_log2_ceil(num_leaves);

	size_t large_block_size = n + num_leaves + padded_num_leaves;

	return alloc_to_device<uintmax_t>(large_block_size);
}

size_t par_find_mismatch(const uintmax_t * const a, const uintmax_t * const b, size_t n) 
{
	uintmax_t *d_a = copy_to_device(a, n);
	uintmax_t *d_large_block = alloc_large_block_to_device(n);

	size_t result = par_find_mismatch(d_a, b, d_large_block, n);

	device_free(d_a);
	device_free(d_large_block);

	return result;
}

__global__ void par_sig_sqrt(const uintmax_t * const p1, uintmax_t *p2, uintmax_t *sig_out, size_t n) {
	size_t num_sqrt = std::sqrt(n - 1) + 1;

	for (auto i = get_tid(); i < n; i += get_num_threads()) {
		p2[i] = p1[i] != p2[i];

		if (p2[i] != 0)
			sig_out[i / num_sqrt] = 1;
	}
}

size_t par_find_mismatch_sa(const uintmax_t * const d_a, const uintmax_t * const b, uintmax_t *d_large_block, size_t n)
{
	uintmax_t *d_b = copy_to_device(d_large_block, b, n);

	uintmax_t *d_section = d_b + n;
	uintmax_t *d_sqrt = d_section + 1;

	par_sig_sqrt<<<BLOCKS, THREADS>>>(d_a, d_b, d_sqrt, n);

	size_t num_sqrt = std::sqrt(n - 1) + 1;

	uintmax_t* h_sqrt = copy_from_device(d_sqrt, num_sqrt);

	size_t section = seq_find_msw(h_sqrt, num_sqrt);

	delete[] h_sqrt;

	if (section == num_sqrt)
		return n;

	size_t section_offset = section * num_sqrt;
	size_t section_end = std::min(n, section_offset + num_sqrt);
	uintmax_t *d_sig_section = d_b + section_offset;
	size_t num_words = section_end - section_offset;

	uintmax_t* h_sig_section = copy_from_device(d_sig_section, num_words);

	size_t result = seq_find_msw(h_sig_section, num_words);

	delete[] h_sig_section;

	return section_offset + result;
}

__global__ void naive_leftmost_prisoner(uintmax_t *p, size_t n)
{
	size_t n_choose_2 = n * (n - 1) / 2;
	size_t num_computations = (n_choose_2 - 1) / get_num_threads() + 1;

	auto i = get_tid() * num_computations;
	size_t b = (1 + std::sqrt(1 + 8 * i)) / 2;
	size_t a = i - b * (b - 1) / 2;

	while (num_computations-- && b < n)
	{
		if (p[a]) p[b] = 0;

		++a;

		if (a == b)
		{
			a = 0;
			++b;
		}
	}
}

size_t par_find_mismatch_s(const uintmax_t * const d_a, const uintmax_t * const b, uintmax_t *d_large_block, size_t n)
{
	uintmax_t *d_b = copy_to_device(d_large_block, b, n);

	uintmax_t *d_section = d_b + n;
	uintmax_t *d_sqrt = d_section + 1;
	size_t num_sqrt = std::sqrt(n - 1) + 1;

	memset_within_device(d_section, 1, 0xFF);

	memset_within_device(d_sqrt, num_sqrt, 0);

	par_sig_sqrt<<<BLOCKS, THREADS>>>(d_a, d_b, d_sqrt, n);

	naive_leftmost_prisoner<<<BLOCKS, THREADS>>>(d_sqrt, num_sqrt);

	par_get_section_diff<<<BLOCKS, THREADS>>>(d_sqrt, d_section, num_sqrt);

	uintmax_t *h_section = copy_from_device(d_section, 1);
	size_t section = *h_section;
	
	if (section == std::numeric_limits<uintmax_t>::max()) {
		delete[] h_section;
		return n;
	}

	memset_within_device(d_section, 1, 0xFF);

	size_t section_offset = section * num_sqrt;
	size_t section_end = std::min(n, section_offset + num_sqrt);
	uintmax_t *d_sig_section = d_b + section_offset;
	size_t num_words = section_end - section_offset;

	naive_leftmost_prisoner<<<BLOCKS, THREADS>>>(d_sig_section, num_words);

	par_get_section_diff<<<BLOCKS, THREADS>>>(d_sig_section, d_section, num_words);

	h_section = copy_from_device(d_section, 1);
	size_t result = *h_section;

	delete[] h_section;

	return section_offset + result;
}

uintmax_t* alloc_large_block_to_device_s(size_t n)
{
	size_t large_block_size = n + std::sqrt(n - 1) + 2;

	return alloc_to_device<uintmax_t>(large_block_size);
}
