#include "hip/hip_runtime.h"
#include "src/BitString.cuh"
#include "src/SkipTrie.hpp"
#include "src/ParallelSkipTrie.cuh"
#include "src/ZipTrie.hpp"
#include "src/ParallelZipTrie.cuh"
#include "src/genetics.cuh"
#include "src/data.hpp"

#include "ctriepp/ctriepp/CTriePP.hpp"
#include "ctriepp/ctriepp/LongString.hpp"

#include <chrono>
#include <algorithm>
#include <random>
#include <string>
#include <vector>
#include <locale.h>

#include <iostream>

using namespace genetics;
using namespace ctriepp;

struct DataPair
{
	Gene gene_bs; // gene represented as BitStrings
	LongString gene_ls; // gene represented as a LongString
};

std::vector<DataPair> load_abchumi_data()
{
	GeneManager gm(ABC_HUMI_DIRECTORY + "ABC-HuMi");

	CPUTimer timer;

	timer.start("Appending ABC-HuMi data");

	auto genes = gm.all_genes();

	static std::vector<std::string> words;
	std::vector<DataPair> data;
	words.reserve(genes.size());
	data.reserve(genes.size());
	for (const auto& gene : genes)
	{
		std::string gene_str;
		for (auto n : gene)
		{
			gene_str += nucleotide_to_char[static_cast<unsigned>(n)];
		}

		words.push_back(gene_str);
		data.push_back({ gene, LongString(&words.back()) });
	}

	timer.print();

	return data;
}

void shuffle_data(std::vector<DataPair>& data, size_t n)
{
	static std::mt19937 gen(std::random_device{}());

	// use fisher-yates to randomize the first n elements of the data
	for (size_t i = 0; i < n; ++i)
	{
		std::uniform_int_distribution<size_t> dist(i, data.size() - 1);
		size_t j = dist(gen);
		std::swap(data[i], data[j]);
	}
}

std::vector<size_t> generate_indices(size_t n)
{
	std::vector<size_t> indices(n);
	std::iota(indices.begin(), indices.end(), 0);
	return indices;
}

void run_construction_benchmark(std::vector<DataPair>& data, size_t n, size_t num_trials, size_t num_repetitions = 100)
{
	static std::random_device rd;
	static std::mt19937 gen(rd());

	CPUTimer timer;

	auto indices = generate_indices(n);

	for (size_t i = 0; i < num_trials; ++i)
	{
		shuffle_data(data, n);
		std::shuffle(indices.begin(), indices.end(), gen);

		size_t max_m = 0;
		size_t N = 0;
		size_t L = 0;

		{
			SkipTrie<Nucleotide, 2> st;

			for (size_t j = 0; j < n; ++j)
			{
				st.insert(&data[j].gene_bs);
			}

			for (size_t j = 0; j < n; ++j)
			{
				const auto& gene = data[j].gene_bs;
				max_m = std::max(max_m, gene.size());
				N += gene.size();
				L += st.lcp_with_others(&gene);
			}
		}

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			CTriePP<bool, false> ctriepp;

			for (size_t j = 0; j < n; ++j)
			{
				ctriepp.insert(data[j].gene_ls, true);
			}
		}

		save_construction_data("c-trie++", n, N, L, timer.elapsed_nanoseconds());

		timer.start();

		// next, test ZipTrie (sequential)
		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ZipTrie<Nucleotide, true, GeometricRank, 2> zt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				zt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("ZT", n, N, L, timer.elapsed_nanoseconds());

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ZipTrie<Nucleotide, false, GeometricRank, 2> mi_zt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				mi_zt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("MI-ZT", n, N, L, timer.elapsed_nanoseconds());

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ParallelZipTrie<Nucleotide, true, GeometricRank, 2> pzt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				pzt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("PZT", n, N, L, timer.elapsed_nanoseconds(), MIN_PAR_COMPARE_WORD_SIZE);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ParallelZipTrie<Nucleotide, false, GeometricRank, 2> mi_pzt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				mi_pzt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("MI-PZT", n, N, L, timer.elapsed_nanoseconds(), MIN_PAR_COMPARE_WORD_SIZE);
	}
}

void run_contains_true_benchmark(std::vector<DataPair>& data, size_t n, size_t num_trials, size_t num_repetitions = 1000)
{
	shuffle_data(data, n);

	size_t max_m = 0;
	for (size_t j = 0; j < n; ++j)
	{
		const auto& gene = data[j].gene_bs;
		max_m = std::max(max_m, gene.size());
	}

	CTriePP<bool, false> ctriepp;
	ZipTrie<Nucleotide, true, GeometricRank, 2> zt(max_m, max_m);
	ZipTrie<Nucleotide, false, GeometricRank, 2> mi_zt(max_m, max_m);
	ParallelZipTrie<Nucleotide, true, GeometricRank, 2> pzt(max_m, max_m);
	ParallelZipTrie<Nucleotide, false, GeometricRank, 2> mi_pzt(max_m, max_m);

	for (size_t j = 0; j < n; ++j)
	{
		ctriepp.insert(data[j].gene_ls, true);
		zt.insert(&data[j].gene_bs);
		mi_zt.insert(&data[j].gene_bs);
		pzt.insert(&data[j].gene_bs);
		mi_pzt.insert(&data[j].gene_bs);
	}

	CPUTimer timer;

	for (size_t i = 0; i < num_trials; ++i)
	{
		auto m = data[i].gene_bs.size();
		auto l = m;

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ctriepp.contains(data[i].gene_ls);
		}

		save_search_data("c-trie++", n, m, l, timer.elapsed_nanoseconds(), num_repetitions);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			zt.contains(&data[i].gene_bs);
		}

		save_search_data("ZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			mi_zt.contains(&data[i].gene_bs);
		}

		save_search_data("MI-ZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			pzt.contains(&data[i].gene_bs);
		}

		save_search_data("PZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, MIN_PAR_COMPARE_WORD_SIZE);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			mi_pzt.contains(&data[i].gene_bs);
		}

		save_search_data("MI-PZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, MIN_PAR_COMPARE_WORD_SIZE);
	}
}

int main(int argc, char* argv[])
{
	setlocale(LC_ALL, "en_US.UTF-8");

	if (argc != 3)
	{
		std::cerr << "Usage: " << argv[0] << " <num_trials> <num_simulations>" << std::endl;
		return 1;
	}

	size_t num_trials = std::stoul(argv[1]);
	size_t num_simulations = std::stoul(argv[2]);

	CPUTimer timer;

	timer.start("Loading ABC-HuMi data");

	auto data = load_abchumi_data();

	timer.print();

	for (size_t n = 1; n <= data.size(); n *= 2)
	{
		for (size_t i = 0; i < num_simulations; ++i)
		{
			timer.start("Running benchmarks for n = " + std::to_string(n) + "(" + std::to_string(i + 1) + "/" + std::to_string(num_simulations) + ")");
			run_construction_benchmark(data, n, num_trials);
			run_contains_true_benchmark(data, n, num_trials);
			timer.print();
		}
	}

	return 0;
}

