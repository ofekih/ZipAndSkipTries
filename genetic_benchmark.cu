#include "hip/hip_runtime.h"
/**
 * @file genetic_benchmark.cu
 * @brief Benchmarking program for trie data structures using genetic data.
 *
 * @details This program benchmarks various trie implementations (SkipTrie, ZipTrie,
 * ParallelSkipTrie, ParallelZipTrie, and C-Trie++) on genetic data from the ABC-HuMi dataset.
 * It measures and compares construction and search performance across different
 * implementations, with a focus on genetic sequence data represented as nucleotides.
 * Results are saved to CSV files for further analysis and visualization.
 *
 * @see synthetic_benchmark.cu
 * @see src/genetics.cuh
 * @see src/BitString.cuh
 * @see src/SkipTrie.hpp
 * @see src/ZipTrie.hpp
 * @see src/ParallelSkipTrie.cuh
 * @see src/ParallelZipTrie.cuh
 */

#include "src/BitString.cuh"
#include "src/SkipTrie.hpp"
#include "src/ParallelSkipTrie.cuh"
#include "src/ZipTrie.hpp"
#include "src/ParallelZipTrie.cuh"
#include "src/genetics.cuh"
#include "src/data.hpp"

#include "ctriepp/ctriepp/CTriePP.hpp"
#include "ctriepp/ctriepp/LongString.hpp"

#include <chrono>
#include <algorithm>
#include <random>
#include <string>
#include <vector>
#include <locale.h>

#include <iostream>

using namespace genetics;
using namespace ctriepp;

/**
 * @struct DataPair
 * @brief A structure holding both BitString and LongString representations of genetic data.
 *
 * @details This structure is used to store the same genetic sequence in two different formats:
 * as a BitString (for our trie implementations) and as a LongString (for C-Trie++),
 * allowing for fair comparison between different data structures.
 */
struct DataPair
{
	Gene gene_bs; ///< Gene represented as BitStrings (for SkipTrie/ZipTrie variants)
	LongString gene_ls; ///< Gene represented as a LongString (for C-Trie++)
};

/**
 * @brief Loads genetic data from the ABC-HuMi dataset.
 *
 * @details Reads all genes from the ABC-HuMi dataset, converts them to both BitString and
 * LongString formats, and returns them as a vector of DataPair objects. The function
 * measures and reports the time taken to load and process the data.
 *
 * @return std::vector<DataPair> A vector containing all genes from the ABC-HuMi dataset
 * in both BitString and LongString formats.
 *
 * @see genetics::GeneManager
 */
std::vector<DataPair> load_abchumi_data()
{
	GeneManager gm(ABC_HUMI_DIRECTORY + "ABC-HuMi");

	CPUTimer timer;

	timer.start("Appending ABC-HuMi data");

	auto genes = gm.all_genes();

	static std::vector<std::string> words;
	std::vector<DataPair> data;
	words.reserve(genes.size());
	data.reserve(genes.size());
	for (const auto& gene : genes)
	{
		std::string gene_str;
		for (auto n : gene)
		{
			gene_str += nucleotide_to_char[static_cast<unsigned>(n)];
		}

		words.push_back(gene_str);
		data.push_back({ gene, LongString(&words.back()) });
	}

	timer.print();

	return data;
}

/**
 * @brief Shuffles the first n elements of the data vector.
 *
 * @details Uses the Fisher-Yates algorithm to randomize the order of the first n elements
 * in the data vector. This is used to create different insertion orders for benchmarking.
 *
 * @param[in,out] data The vector of DataPair objects to shuffle.
 * @param n The number of elements to shuffle.
 */
void shuffle_data(std::vector<DataPair>& data, size_t n)
{
	static std::mt19937 gen(std::random_device{}());

	// use fisher-yates to randomize the first n elements of the data
	for (size_t i = 0; i < n; ++i)
	{
		std::uniform_int_distribution<size_t> dist(i, data.size() - 1);
		size_t j = dist(gen);
		std::swap(data[i], data[j]);
	}
}

/**
 * @brief Generates a vector of indices from 0 to n-1.
 *
 * @details Creates a vector containing integers from 0 to n-1 in ascending order.
 * This is used for creating random access patterns for search benchmarks.
 *
 * @param n The number of indices to generate.
 * @return std::vector<size_t> A vector containing integers from 0 to n-1.
 */
std::vector<size_t> generate_indices(size_t n)
{
	std::vector<size_t> indices(n);
	std::iota(indices.begin(), indices.end(), 0);
	return indices;
}

/**
 * @brief Runs construction benchmarks on various trie implementations using genetic data.
 *
 * @details Measures the time taken to construct different trie data structures (C-Trie++,
 * ZipTrie, memory-intensive ZipTrie, ParallelZipTrie, memory-intensive ParallelZipTrie,
 * SkipTrie, and ParallelSkipTrie) with the same genetic data. For each implementation,
 * the benchmark is repeated multiple times and the results are saved to CSV files.
 *
 * @param[in,out] data The vector of DataPair objects containing the genetic data.
 * @param n The number of genes to use in the benchmark.
 * @param num_trials The number of trials to run (each with a different shuffle).
 * @param num_repetitions The number of repetitions for each trial (default: 100).
 *
 * @see save_construction_data
 */
void run_construction_benchmark(std::vector<DataPair>& data, size_t n, size_t num_trials, size_t num_repetitions = 100)
{
	static std::random_device rd;
	static std::mt19937 gen(rd());

	CPUTimer timer;

	auto indices = generate_indices(n);

	for (size_t i = 0; i < num_trials; ++i)
	{
		shuffle_data(data, n);
		std::shuffle(indices.begin(), indices.end(), gen);

		size_t max_m = 0;
		size_t N = 0;
		size_t L = 0;

		{
			SkipTrie<Nucleotide, 2> st;

			for (size_t j = 0; j < n; ++j)
			{
				st.insert(&data[j].gene_bs);
			}

			for (size_t j = 0; j < n; ++j)
			{
				const auto& gene = data[j].gene_bs;
				max_m = std::max(max_m, gene.size());
				N += gene.size();
				L += st.lcp_with_others(&gene);
			}
		}

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			CTriePP<bool, false> ctriepp;

			for (size_t j = 0; j < n; ++j)
			{
				ctriepp.insert(data[j].gene_ls, true);
			}
		}

		save_construction_data("c-trie++", n, N, L, timer.elapsed_nanoseconds(), 1, 0, true);

		timer.start();

		// next, test ZipTrie (sequential)
		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ZipTrie<Nucleotide, true, GeometricRank, 2> zt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				zt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("ZT", n, N, L, timer.elapsed_nanoseconds(), 1, 0, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ZipTrie<Nucleotide, false, GeometricRank, 2> mi_zt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				mi_zt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("MI-ZT", n, N, L, timer.elapsed_nanoseconds(), 1, 0, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ParallelZipTrie<Nucleotide, true, GeometricRank, 2> pzt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				pzt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("PZT", n, N, L, timer.elapsed_nanoseconds(), 1, MIN_PAR_COMPARE_WORD_SIZE, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ParallelZipTrie<Nucleotide, false, GeometricRank, 2> mi_pzt(max_m, max_m);

			for (size_t j = 0; j < n; ++j)
			{
				mi_pzt.insert(&data[j].gene_bs);
			}
		}

		save_construction_data("MI-PZT", n, N, L, timer.elapsed_nanoseconds(), 1, MIN_PAR_COMPARE_WORD_SIZE, true);
	}
}

/**
 * @brief Runs search benchmarks on various trie implementations using genetic data.
 *
 * @details Measures the time taken to search for genes in different trie data structures
 * (C-Trie++, ZipTrie, memory-intensive ZipTrie, ParallelZipTrie, memory-intensive ParallelZipTrie)
 * with the same genetic data. For each implementation, the benchmark is repeated multiple times
 * with different search patterns, and the results are saved to CSV files.
 *
 * @param[in,out] data The vector of DataPair objects containing the genetic data.
 * @param n The number of genes to use in the benchmark.
 * @param num_trials The number of trials to run (each with a different search pattern).
 * @param num_repetitions The number of repetitions for each trial (default: 1000).
 *
 * @see save_search_data
 */
void run_contains_true_benchmark(std::vector<DataPair>& data, size_t n, size_t num_trials, size_t num_repetitions = 1000)
{
	shuffle_data(data, n);

	size_t max_m = 0;
	for (size_t j = 0; j < n; ++j)
	{
		const auto& gene = data[j].gene_bs;
		max_m = std::max(max_m, gene.size());
	}

	CTriePP<bool, false> ctriepp;
	ZipTrie<Nucleotide, true, GeometricRank, 2> zt(max_m, max_m);
	ZipTrie<Nucleotide, false, GeometricRank, 2> mi_zt(max_m, max_m);
	ParallelZipTrie<Nucleotide, true, GeometricRank, 2> pzt(max_m, max_m);
	ParallelZipTrie<Nucleotide, false, GeometricRank, 2> mi_pzt(max_m, max_m);

	for (size_t j = 0; j < n; ++j)
	{
		ctriepp.insert(data[j].gene_ls, true);
		zt.insert(&data[j].gene_bs);
		mi_zt.insert(&data[j].gene_bs);
		pzt.insert(&data[j].gene_bs);
		mi_pzt.insert(&data[j].gene_bs);
	}

	CPUTimer timer;

	for (size_t i = 0; i < num_trials; ++i)
	{
		auto m = data[i].gene_bs.size();
		auto l = m;

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			ctriepp.contains(data[i].gene_ls);
		}

		save_search_data("c-trie++", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, 0, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			zt.contains(&data[i].gene_bs);
		}

		save_search_data("ZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, 0, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			mi_zt.contains(&data[i].gene_bs);
		}

		save_search_data("MI-ZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, 0, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			pzt.contains(&data[i].gene_bs);
		}

		save_search_data("PZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, MIN_PAR_COMPARE_WORD_SIZE, true);

		timer.start();

		for (size_t _ = 0; _ < num_repetitions; ++_)
		{
			mi_pzt.contains(&data[i].gene_bs);
		}

		save_search_data("MI-PZT", n, m, l, timer.elapsed_nanoseconds(), num_repetitions, MIN_PAR_COMPARE_WORD_SIZE, true);
	}
}

/**
 * @brief Main function that runs the genetic benchmarks.
 *
 * @details Loads genetic data from the ABC-HuMi dataset and runs construction and search
 * benchmarks on various trie implementations. The benchmark parameters (number of trials
 * and number of simulations) can be specified as command-line arguments.
 *
 * @param argc Number of command-line arguments.
 * @param argv Array of command-line argument strings.
 * @return int Returns 0 on successful execution, 1 on invalid arguments.
 */
int main(int argc, char* argv[])
{
	setlocale(LC_ALL, "en_US.UTF-8");

	if (argc != 3)
	{
		std::cerr << "Usage: " << argv[0] << " <num_trials> <num_simulations>" << std::endl;
		return 1;
	}

	size_t num_trials = std::stoul(argv[1]);
	size_t num_simulations = std::stoul(argv[2]);

	CPUTimer timer;

	timer.start("Loading ABC-HuMi data");

	auto data = load_abchumi_data();

	timer.print();

	for (size_t n = 1; n <= data.size(); n *= 2)
	{
		for (size_t i = 0; i < num_simulations; ++i)
		{
			timer.start("Running benchmarks for n = " + std::to_string(n) + "(" + std::to_string(i + 1) + "/" + std::to_string(num_simulations) + ")");
			run_construction_benchmark(data, n, num_trials);
			run_contains_true_benchmark(data, n, num_trials);
			timer.print();
		}
	}

	return 0;
}

