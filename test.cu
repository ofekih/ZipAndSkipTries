#include "hip/hip_runtime.h"
#include "src/BitString.cuh"
#include "src/SkipTrie.hpp"
#include "src/ParallelSkipTrie.cuh"
#include "src/synthetic.hpp"
// #include "src/ZipZipTrie.hpp"
#include "src/ZipTrie.hpp"
#include "src/ParallelZipTrie.cuh"

#include <chrono>
#include <algorithm>
#include <random>
#include <string>
#include <vector>

#include <iostream>

const std::vector<std::string> WORDS = {
	"ANT", "APPLE", "APPLY", "APT", "APTLY", "AQUA", "PAL", "PEACE", "PEACH", "PEAK", "PENGUIN", "PALA", "AAAAAAAAAAAAAAAAAA", "AAAAAAAAAAAAAAAAAAVB"
};

BitString<char, 64> from_string(const std::string& str)
{
	return BitString<char, 64>(str);
}

// ZipZipTrie<char, false> create_test_zzt()
// {
// 	static std::unordered_map<std::string, BitString<char>> word_to_bs;
// 	static std::unordered_map<std::string, unsigned> word_to_index;
	
// 	if (word_to_bs.empty())
// 	{
// 		for (unsigned i = 0; i < WORDS.size(); ++i)
// 		{
// 			word_to_bs[WORDS[i]] = from_string(WORDS[i]);
// 			word_to_index[WORDS[i]] = i;
// 		}
// 	}

// 	unsigned longest_word_length = std::max_element(WORDS.begin(), WORDS.end(), [](const std::string& a, const std::string& b) { return a.size() < b.size(); })->size();

// 	ZipZipTrie<char, false> trie(WORDS.size(), longest_word_length);
// 	const unsigned NULLPTR = std::numeric_limits<unsigned>::max();

// 	trie.set_root_index(word_to_index["ANT"]);

// 	trie.set(&word_to_bs["ANT"], { 3, 52 }, NULLPTR, word_to_index["APPLY"], 0, 0);
// 	trie.set(&word_to_bs["APPLE"], { 2, 12 }, NULLPTR, NULLPTR, 4, 1);
// 	trie.set(&word_to_bs["APPLY"], { 3, 21 }, word_to_index["APPLE"], word_to_index["APT"], 1, 0);
// 	trie.set(&word_to_bs["APT"], { 3, 3 }, NULLPTR, word_to_index["PEAK"], 2, 0);
// 	trie.set(&word_to_bs["APTLY"], { 0, 45 }, NULLPTR, NULLPTR, 1, 3);
// 	trie.set(&word_to_bs["AQUA"], { 2, 0 }, word_to_index["APTLY"], word_to_index["PEACE"], 0, 1);
// 	trie.set(&word_to_bs["PAL"], { 0, 33 }, NULLPTR, NULLPTR, 1, 0);
// 	trie.set(&word_to_bs["PEACE"], { 1, 8 }, word_to_index["PAL"], word_to_index["PEACH"], 0, 3);
// 	trie.set(&word_to_bs["PEACH"], { 1, 2 }, NULLPTR, NULLPTR, 4, 3);
// 	trie.set(&word_to_bs["PEAK"], { 2, 9 }, word_to_index["AQUA"], word_to_index["PENGUIN"], 0, 0);
// 	trie.set(&word_to_bs["PENGUIN"], { 0, 1 }, NULLPTR, NULLPTR, 2, 0);

// 	return trie;
// }

int main()
{
	// auto word = get_random_word(100);
	// std::cout << word << std::endl;
	std::vector<BitString<char, 64>> bit_strings;
	std::transform(WORDS.begin(), WORDS.end(), std::back_inserter(bit_strings), from_string);

	unsigned longest_word_length = std::max_element(WORDS.begin(), WORDS.end(), [](const std::string& a, const std::string& b) { return a.size() < b.size(); })->size();

	ZipTrie<char, true, GeometricRank, 64> trie(WORDS.size(), longest_word_length);
	// ParallelZipTrie<char, false, GeometricRank, 64> trie(WORDS.size(), longest_word_length);
	// ParallelSkipTrie<char, 64> trie(longest_word_length);
	// ZipZipTrie<char, unsigned> trie = create_test_zzt();

	// if (!trie.contains(&bit_strings[4]))
	// {
	// 	std::cout << "Word " << bit_strings[4].to_string() << " not found in trie" << std::endl;
	// }


	trie.insert(&bit_strings[0]);
	trie.insert(&bit_strings[1]);
	trie.insert(&bit_strings[2]);
	trie.insert(&bit_strings[3]);
	trie.insert(&bit_strings[4]);
	trie.insert(&bit_strings[5]);
	trie.insert(&bit_strings[6]);
	trie.insert(&bit_strings[7]);
	trie.insert(&bit_strings[8]);
	trie.insert(&bit_strings[9]);
	trie.insert(&bit_strings[10]);
	trie.insert(&bit_strings[11]);
	// trie.insert(&bit_strings[12]);
	trie.insert(&bit_strings[13]);

	for (const auto& word : bit_strings)
	{
		if (!trie.contains(&word))
		{
			// should printf something like: <word> not found in trie, but shares an LCP length of <lcp_length>
			// use trie.lcp(&word) to get the LCP length
			printf("%s not found in trie, but shares an LCP length of %u\n", word.to_string().c_str(), trie.lcp(&word));
		}
	}

	trie.to_dot("test.dot");

	// use system to run dot -Tpng test.dot -o test.png
	int result = system("dot -Tpng test.dot -o test.png");
	if (result != 0) {
		std::cerr << "Error: Failed to generate PNG from DOT file. Return code: " << result << std::endl;
		return 1;
	}

	// SkipTrie<char> trie;
	// for (const auto& bit_string : bit_strings)
	// {
	// 	printf("Inserting %s\n", bit_string.to_string().c_str());
	// 	trie.insert(&bit_string);
	// 	std::cout << trie << std::endl;

	// 	printf("\n\n");
	// }

	return 0;
}
